
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>

#define N 50000
#define CORES 50000

/* run the collatz conjecture and return the number of steps */
__global__ void collatz(unsigned int* step) {
    //Set x to the initial value of step
    unsigned int x = step[blockIdx.x];
    //Reset step to 0
    step[blockIdx.x] = 0;

    /* do the iterative process */
    while (x != 1) {
        if ((x % 2) == 0) {
            x = x / 2;
        } else {
            x = 3 * x + 1;
        }
        step[blockIdx.x]++;
    }
}

int main( ) {
    /* store the number of steps for each number up to N */
    unsigned int cpu_steps[N];
    unsigned int* gpu_steps;
    
    /* allocate space on the GPU */
    hipMalloc((void**) &gpu_steps, N * sizeof(unsigned int));
    
    for(int i=0; i < N; i++) {
        cpu_steps[i] = i+1;
    }

    /* send gpu_steps to the GPU */
    hipMemcpy(gpu_steps, cpu_steps, N * sizeof(unsigned int), hipMemcpyHostToDevice);
    
    /* run the collatz conjecture on all N items */
    collatz<<<CORES, 1>>>(gpu_steps);
    
    /* send gpu_steps back to the CPU */
    hipMemcpy(cpu_steps, gpu_steps, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    /* free the memory on the GPU */
    hipFree(gpu_steps);

    /* find the largest */
    unsigned int largest = cpu_steps[0], largest_i = 0;
    for (int i = 1; i < N; i++) {
        if (cpu_steps[i] > largest) {
            largest = cpu_steps[i];
            largest_i = i;
        }
    }

    /* report results */
    printf("The longest collatz chain up to %d is %d with %d steps.\n",
            N, largest_i + 1, largest);

    return 0;
}

